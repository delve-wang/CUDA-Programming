
#include <hip/hip_runtime.h>
#include <stdlib.h> // malloc() and free()
#include <stdio.h> // printf()
double get_length(double *x, int N);

int main(void)
{
    int N = 1000;
    int M = sizeof(double) * N;
    double *x = (double *) malloc(M);
    for (int n = 0; n < N; ++n)
    {
        x[n] = 1.0;
    }
    double length = get_length(x, N);
    printf("length = %g.\n", length);
    free(x);
    return 0;
}

void __global__ get_length(double *g_x, double *g_length, int N)
{
    int tid = threadIdx.x;
    __shared__ double s_x[1024];
    s_x[tid] = 0.0;
    if (tid < N)
    {
        double tmp = g_x[tid];
        s_x[tid] = tmp * tmp;
    }
    __syncthreads();
    for (int offset = blockDim.x / 2; offset > 0; offset /= 2)
    {
        if (tid < offset)
        {
            s_x[tid] += s_x[tid + offset];
        }
        __syncthreads();
    }
    if (tid == 0)
    {
        g_length[0] = sqrt(s_x[0]);
    }
}

double get_length(double *x, int N)
{
    double *g_length;
    hipMalloc((void**)&g_length, sizeof(double));
    double *g_x;
    hipMalloc((void**)&g_x, sizeof(double) * N);
    hipMemcpy(g_x, x, sizeof(double) * N, hipMemcpyHostToDevice);
    get_length<<<1, 1024>>>(g_x, g_length, N);
    double *cpu_length = (double *) malloc(sizeof(double));
    hipMemcpy(cpu_length, g_length, sizeof(double), hipMemcpyDeviceToHost);
    hipFree(g_length);
    hipFree(g_x);
    double length = cpu_length[0];
    free(cpu_length);
    return length;
}

